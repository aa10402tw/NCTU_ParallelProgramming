
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

// Kernel function
__global__ void VecAdd(float* A, float* B, float* C) 
{
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
}

int main() 
{
	int N = 10;

	// Allocate CPU Memory
	float* A = (float*)malloc(N * sizeof(float));
	float* B = (float*)malloc(N * sizeof(float));
	float* C = (float*)malloc(N * sizeof(float));

	// Allocate GPU Memory
	float *gpu_A, *gpu_B, *gpu_C;
	hipMalloc(&gpu_A, N * sizeof(float));
	hipMalloc(&gpu_B, N * sizeof(float));
	hipMalloc(&gpu_C, N * sizeof(float));

	// Init value
	for (int i = 0; i < N; i++) {
		A[i] = i;
		B[i] = 2 * i;
		C[i] = 0;
	}

	// Copy Data from CPU memory to GPU memory
	hipMemcpy(gpu_A, A, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(gpu_B, B, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(gpu_C, C, N * sizeof(float), hipMemcpyHostToDevice);

	// GPU function (launch kernel function)
	VecAdd <<<1, N>>> (gpu_A, gpu_B, gpu_C);

	// Copy Data from GPU memory to CPU memory
	hipMemcpy(A, gpu_A, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(B, gpu_B, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(C, gpu_C, N * sizeof(float), hipMemcpyDeviceToHost);

	// Print Result
	for (int i = 0; i < N; i++) {
		printf("%f,", C[i]);
	}
	return 0;
}