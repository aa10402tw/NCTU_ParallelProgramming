/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update(void);
void printfinal(void);


/**********************************************************************
 *  Checks input values from parameters
 *********************************************************************/
void check_param(int nsteps, int tpoints)
{
	char tchar[20];

	/* check number of points, number of iterations */
	while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
		printf("Enter number of points along vibrating string [%d-%d]: "
			, MINPOINTS, MAXPOINTS);
		scanf("%s", tchar);
		tpoints = atoi(tchar);
		if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
			printf("Invalid. Please enter value between %d and %d\n",
				MINPOINTS, MAXPOINTS);
	}
	while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
		printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
		scanf("%s", tchar);
		nsteps = atoi(tchar);
		if ((nsteps < 1) || (nsteps > MAXSTEPS))
			printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
	}

	printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(float* oldval, float* values, int tpoints)
{
	int i, j;
	float x, fac, k, tmp;

	/* Calculate initial values based on sine curve */
	fac = 2.0 * PI;
	k = 0.0;
	tmp = tpoints - 1;
	for (j = 1; j <= tpoints; j++) {
		x = k / tmp;
		values[j] = sin(fac * x);
		k = k + 1.0;
	}

	/* Initialize old values array */
	for (i = 1; i <= tpoints; i++)
		oldval[i] = values[i];
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
void do_math(float* oldval, float* values, float* newval, int i)
{
	float dtime, c, dx, tau, sqtau;

	dtime = 0.3;
	c = 1.0;
	dx = 1.0;
	tau = (c * dtime / dx);
	sqtau = tau * tau;
	newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
void update(float* oldval, float* values, float* newval, int nsteps, int tpoints)
{
	int i, j;

	/* Update values for each time step */
	for (i = 1; i <= nsteps; i++) {
		/* Update points along line for this time step */
		for (j = 1; j <= tpoints; j++) {
			/* global endpoints */
			if ((j == 1) || (j == tpoints))
				newval[j] = 0.0;
			else
				do_math(oldval, values, newval, j);
		}

		/* Update old values with new values */
		for (j = 1; j <= tpoints; j++) {
			oldval[j] = values[j];
			values[j] = newval[j];
		}
	}
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal(float* values, int tpoints)
{
	int i;

	for (i = 1; i <= tpoints; i++) {
		printf("%6.4f ", values[i]);
		if (i % 10 == 0)
			printf("\n");
	}
}

__global__ void init_line_kernel(float* oldval, float* values, int tpoints) {
	//int i, j;
	float x, fac, k, tmp;
	
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x; // Total number of threads
	if (tid == 0) {
		printf("gridDim   (%d, %d, %d)\n", gridDim.x, gridDim.y, gridDim.z);
		printf("blockDim   (%d, %d, %d)\n", blockDim.x, blockDim.y, blockDim.z);
	}
	
	fac = 2.0 * PI;
	tmp = (float)(tpoints - 1);

	for (int idx = tid; idx <= tpoints; idx += offset) {
		if (idx >= 1) {
			k = (float)(idx - 1);
			x = k / tmp;
			values[idx] = __sinf(fac * x);
			oldval[idx] = values[idx];
		}	
	}
}

__global__ void update_kernel(float* oldval, float* values, float* newval, int nsteps, int tpoints)
{
	int i, j;
	int tid = blockIdx.x * blockDim.x + threadIdx.x; // threadId
	int offset = blockDim.x * gridDim.x; // Total number of threads

	/* Update values for each time step */
	for (i = 1; i <= nsteps; i++) {
		for (int j = tid; j <= tpoints; j += offset) {
			/* global endpoints */
			if ((j == 1) || (j == tpoints))
				newval[j] = 0.0;
			else {
				float dtime, c, dx, tau, sqtau;
				dtime = 0.3;
				c = 1.0;
				dx = 1.0;
				tau = (c * dtime / dx);
				sqtau = tau * tau;
				newval[j] = (2.0 * values[j]) - oldval[j] + (sqtau *  (-2.0)*values[j]);
			}
			/* Update old values with new values */
			oldval[j] = values[j];
			values[j] = newval[j];
		}
	}
}
void checkIsSame(float* A, float* B, int n) {
	for (int i = 0; i < n; i++) {
		// printf("%d : (%f, %f) [%f]\n", i, A[i], B[i], A[i]-B[i]);
		if ( A[i] - B[i] > 0.0001 || A[i] - B[i] < -0.0001) {
			printf("\n\nDifferent at %d (%f v.s %f)\n", i, A[i], B[i]);
			return;
		}
			
	}
	printf("\n\nIs Same\n");
}

/**********************************************************************
 *  Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	int nsteps,                     /* number of time steps */
		tpoints,					/* total points along string */
		rcode;                      /* generic return code */

	sscanf(argv[1], "%d", &tpoints);
	sscanf(argv[2], "%d", &nsteps);
	check_param(nsteps, tpoints);

	int threadsPerBlock = 512;
	int numBlocks = (tpoints / threadsPerBlock) + 1;

	threadsPerBlock = 10;
	numBlocks = 1;

	float *oldval, *values, *newval;
	float *cpu_val, *gpu_val;

	/******************/
	/* Initialization */
	/******************/
	// CPU
	printf("\n\n--- [CPU Version Init] ---\n");
	oldval = (float*)malloc((MAXPOINTS + 2) * sizeof(float)); /* values at time (t-dt) */
	values = (float*)malloc((MAXPOINTS + 2) * sizeof(float)); /* values at time t */
	newval = (float*)malloc((MAXPOINTS + 2) * sizeof(float)); /* values at time (t+dt) */
	init_line(oldval, values, tpoints);
	//printfinal(values, tpoints);

	// Debug
	cpu_val = (float*)malloc((MAXPOINTS + 2) * sizeof(float));
	for (int i = 0; i <= tpoints; i++)
		cpu_val[i] = values[i];

	// GPU 
	printf("\n\n--- [GPU Version Init] ---\n");
	oldval = (float*)malloc((MAXPOINTS + 2) * sizeof(float)); /* values at time (t-dt) */
	values = (float*)malloc((MAXPOINTS + 2) * sizeof(float)); /* values at time t */
	newval = (float*)malloc((MAXPOINTS + 2) * sizeof(float)); /* values at time (t+dt) */

	float *gpu_oldval, *gpu_values, *gpu_newval;
	hipMalloc(&gpu_oldval, (MAXPOINTS + 2) * sizeof(float));
	hipMalloc(&gpu_values, (MAXPOINTS + 2) * sizeof(float));
	hipMalloc(&gpu_newval, (MAXPOINTS + 2) * sizeof(float));

	init_line_kernel <<<numBlocks, threadsPerBlock>>> (gpu_oldval, gpu_values, tpoints);

	hipMemcpy(oldval, gpu_oldval, (MAXPOINTS + 2) * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(values, gpu_values, (MAXPOINTS + 2) * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(newval, gpu_newval, (MAXPOINTS + 2) * sizeof(float), hipMemcpyDeviceToHost);
	// printfinal(values, tpoints);

	// Debug
	gpu_val = (float*)malloc((MAXPOINTS + 2) * sizeof(float));
	for (int i = 0; i <= tpoints; i++)
		gpu_val[i] = values[i];

	// Check 
	checkIsSame(cpu_val, gpu_val, tpoints+1);



	/**********/
	/* Update */
	/**********/

	// CPU
	printf("\n\n--- [CPU Version Update] ---\n");
	oldval = (float*)malloc((MAXPOINTS + 2) * sizeof(float)); /* values at time (t-dt) */
	values = (float*)malloc((MAXPOINTS + 2) * sizeof(float)); /* values at time t */
	newval = (float*)malloc((MAXPOINTS + 2) * sizeof(float)); /* values at time (t+dt) */
	init_line(oldval, values, tpoints);
	update(oldval, values, newval, nsteps, tpoints);
	//printfinal(values, tpoints);

	// Debug
	cpu_val = (float*)malloc((MAXPOINTS + 2) * sizeof(float));
	for (int i = 0; i <= tpoints; i++)
		cpu_val[i] = values[i];

	// GPU 
	printf("\n\n--- [GPU Version Update] ---\n");
	oldval = (float*)malloc((MAXPOINTS + 2) * sizeof(float)); /* values at time (t-dt) */
	values = (float*)malloc((MAXPOINTS + 2) * sizeof(float)); /* values at time t */
	newval = (float*)malloc((MAXPOINTS + 2) * sizeof(float)); /* values at time (t+dt) */

	hipMalloc(&gpu_oldval, (MAXPOINTS + 2) * sizeof(float));
	hipMalloc(&gpu_values, (MAXPOINTS + 2) * sizeof(float));
	hipMalloc(&gpu_newval, (MAXPOINTS + 2) * sizeof(float));

	//int threadsPerBlock = 10;
	//int numBlocks = (tpoints / threadsPerBlock) + 1;
	init_line_kernel <<<numBlocks, threadsPerBlock>>> (gpu_oldval, gpu_values, tpoints);
	update_kernel <<<numBlocks, threadsPerBlock >>> (gpu_oldval, gpu_values, gpu_newval, nsteps, tpoints);

	hipMemcpy(oldval, gpu_oldval, (MAXPOINTS + 2) * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(values, gpu_values, (MAXPOINTS + 2) * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(newval, gpu_newval, (MAXPOINTS + 2) * sizeof(float), hipMemcpyDeviceToHost);
	//printfinal(values, tpoints);

	// Debug
	gpu_val = (float*)malloc((MAXPOINTS + 2) * sizeof(float));
	for (int i = 0; i <= tpoints; i++)
		gpu_val[i] = values[i];

	// Check 
	checkIsSame(cpu_val, gpu_val, tpoints + 1);


	return 0;
}